
#include <hip/hip_runtime.h>
extern "C"
__global__ void add_strided_float(int n, float *dx, float *dy) {
       int  dxIdx = blockDim.x * blockIdx.x + threadIdx.x;
          int  incx = blockDim.x * gridDim.x;
          int incy = blockDim.y * gridDim.y;
          int dyIdx = blockDim.y * gridDim.y + threadIdx.y;

      for (int c = 0, xi = dxIdx, yi = dyIdx; c < n; c++, xi += incx, yi += incy) {
                         dy[yi] += dx[xi];
       }
 }


